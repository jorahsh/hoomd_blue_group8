#include "hip/hip_runtime.h"
// Copyright (c) 2009-2016 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "DyBondUpdater.cuh"

/*! \file DyBondUpdater.cu
    \brief CUDA kernels for DyBondUpdater
*/

// First, the kernel code for zeroing the velocities on the GPU
//! Kernel that zeroes velocities on the GPU
/*! \param d_vel Velocity-mass array from the ParticleData
    \param N Number of particles

    This kernel executes one thread per particle and zeros the velocity of each. It can be run with any 1D block size
    as long as block_size * num_blocks is >= the number of particles.
*/
extern "C" __global__
void gpu_zero_velocities_kernel(volatile bool *found,
                                Scalar4 *d_pos,
                                unsigned int N, 
                                const group_storage<2> *blist,
                                const unsigned int pitch,
                                const unsigned int *n_bonds_list,
                                const unsigned int n_bond_type,
                                const unsigned int *d_n_neigh,
                                const unsigned int *d_nlist,
                                const unsigned int *d_head_list,
                                const unsigned int size_nlist)
    {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("in cuda kernel. blockIdx.x:%d,blockDim.x:%d,threadIdx.x:%d\n",blockIdx.x,blockDim.x,threadIdx.x);
    // load in the length of the list for this thread (MEM TRANSFER: 4 bytes)
    if (idx < N)
        {
        // read in the position of our b-particle from the a-b-c triplet. (MEM TRANSFER: 16 bytes)
        Scalar4 idx_postype = d_pos[idx];  // we can be either a, b, or c in the a-b-c-d quartet
        unsigned int bond_from_type = __scalar_as_int(idx_postype.w);
        if (bond_from_type != 2)
	    {
            int bond_to_type = (bond_from_type==0)?1:0;
            // load in the length of the list
    	    unsigned int n_neigh = d_n_neigh[idx];
    	    const unsigned int head_idx = d_head_list[idx];
            //printf("particle id:%d, n_neigh:%d, head_idx:%d\n",idx,n_neigh,head_idx);

            unsigned int next_neigh(0);
            // loop over neighbors
            for (int neigh_idx = 0; neigh_idx < n_neigh; neigh_idx++)
                 {
                 next_neigh = d_nlist[head_idx + neigh_idx + 1];
                 Scalar4 idx_postype = d_pos[next_neigh];  // we can be either a, b, or c in the a-b-c-d quartet
                 unsigned int ptype = __scalar_as_int(idx_postype.w);
                 if (ptype==bond_to_type)
                     {
                     printf("Bond from %d(%d) to %d(%d)\n",idx,bond_from_type,next_neigh,bond_to_type);
                     // MEM TRANSFER: 8 bytes
                     group_storage<2> cur_bond = blist[0];
                     printf("cur_bond %d,%d\n",cur_bond.idx[0],cur_bond.idx[1]);
                     }
                 }
            }        
        }
        
    }

/*! \param d_vel Velocity-mass array from the ParticleData
    \param N Number of particles
    This is just a driver for gpu_zero_velocities_kernel(), see it for the details
*/
hipError_t gpu_zero_velocities(Scalar4 *d_pos, 
				unsigned int N,
				const group_storage<2> *blist,
                const unsigned int pitch,
                const unsigned int *n_bonds_list,
                const unsigned int n_bond_type,
                const unsigned int *d_n_neigh,
                const unsigned int *d_nlist,
                const unsigned int *d_head_list,
                const unsigned int size_nlist)
    {
    // setup the grid to run the kernel
    int block_size = 256;
    dim3 grid( (int)ceil((double)N / (double)block_size), 1, 1);
    dim3 threads(block_size, 1, 1);
    volatile bool found = false;
    bool *foundptr = (bool*) &found;
    // run the kernel
    printf("Going to call the kernel\n");
    gpu_zero_velocities_kernel<<< grid, threads >>>(foundptr,
                                                    d_pos,
                                                    N,
                                                    blist,
                                                    pitch,
                                                    n_bonds_list,
                                                    n_bond_type,
						    d_n_neigh,
						    d_nlist,
						    d_head_list,
						    size_nlist);

    // this method always succeds. If you had a cuda* call in this driver, you could return its error code if not
    // hipSuccess
    return hipSuccess;
    }
